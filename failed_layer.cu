#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>
#include <iostream>
#include <vector>
#include <fstream>
#include <random>
#include <algorithm>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "nvtx3/nvToolsExt.h"

using namespace std;

typedef struct block {
	// thrust::device_vector
	thrust::device_vector<int> offset;
	thrust::device_vector<int> indices;
	thrust::device_vector<int> unique;

	void clear(){
		offset.clear();
		indices.clear();
	}
} block;

typedef struct graphStruct {
	thrust::device_vector<int> indptr;
	thrust::device_vector<int> indices;
} graphStruct;


void remove_duplicates(thrust::device_vector<int>& nodes){
  if(nodes.size() == 0)return;
  if(nodes.size() > 1){
    nvtxRangePush("remove duplicates"); // built in timing?
    thrust::sort(nodes.begin(), nodes.end());
    auto it = thrust::unique(nodes.begin(), nodes.end());
    nodes.erase(it, nodes.end());
    nvtxRangePop();
  }
}

//probably not parallelized
__global__ void sample_layer(struct graphStruct* graph, struct block* t_block, thrust::device_vector<int> target) {
	int offset = 0;
	//parallelize this for loop
	for (int x: target) {
		t_block->offset.push_back(offset);
		for (int i = graph->indptr[x]; i < graph->indptr[x+1]; ++i, ++offset) {
			t_block->indices.push_back(graph->indices[i]);
		}
	} t_block->offset.push_back(offset);
	t_block->unique = t_block->indices;
}

int main() {
	// reading the file shouldn't be just move the data read into device
	fstream f("../data/graph", ios::in);
	int num_ptrs;
	int num_edges;
	int num_sample;
	f >> num_ptrs;
	f >> num_edges;
	f >> num_sample;

	// try these diff types
	// malloc
	// mallocmanaged
	// mallocHostAlloc

	fstream nodesf("../data/indptr", ios::in | ios::binary );
	if(!nodesf) {
		cout << "cannot open file!\n";
		return 0;
	}
	long *nodes_h = (long *)malloc (num_ptrs * sizeof(long));
	long *nodes_d;
	hipMalloc((void**) &nodes_d, ((num_ptrs + 1) * sizeof(long)));
	nodesf.read((char *)nodes_h, (num_ptrs * sizeof(long)));
	hipMemcpy(nodes_d, nodes_h, (num_ptrs + 1) * sizeof(long) , hipMemcpyHostToDevice);

	fstream edgesf("../data/indices", ios::in | ios::binary );
	if(!edgesf) {
		cout << "cannot open file!\n";
		return 0;
	}
	long *edges_h = (long *)malloc (num_edges * sizeof(long));
	long *edges_d;
	hipMalloc((void**) &edges_d, ((num_edges + 1) * sizeof(long)));
	edgesf.read((char *)edges_h, (num_edges * sizeof(long)));
	hipMemcpy(edges_d, edges_h, (num_ptrs + 1) * sizeof(long) , hipMemcpyHostToDevice);

	fstream samplef("../data/train", ios::in | ios::binary );
	if(!samplef) {
		cout << "cannot open file!\n";
		return 0;
	}
	long *sample_h = (long *)malloc (num_sample * sizeof(long));
	long *sample_d;
	hipMalloc((void**) &sample_d, ((num_sample + 1) * sizeof(long)));
	samplef.read((char *)sample_h, (num_sample * sizeof(long)));
	hipMemcpy(sample_d, sample_h, (num_ptrs + 1) * sizeof(long) , hipMemcpyHostToDevice);

	// not sure how to convert batching to gpu
	vector<vector<int>> batches;

	for (int i = 0; i < num_sample - 1024; i += 1024) {
		vector<int> batch;
		for (int j = i; j < i + 1024; ++j) {
			batch.push_back(sample_h[j]);
		}
		batches.push_back(batch);
	}
	// end

	vector<int> a(nodes_h, nodes_h + num_ptrs);
	vector<int> b(edges_h, edges_h + num_edges);
	graphStruct h_graph = {a, b};
	graphStruct *d_graph;

	hipMalloc((void **) &d_graph, sizeof(h_graph));
	hipMemcpy(d_graph, &h_graph, sizeof(h_graph), hipMemcpyHostToDevice);


	block arr[batches.size()];
	block *d_arr;

	hipMalloc((void **) &d_arr, sizeof(arr));
	hipMemcpy(d_arr, &arr, sizeof(arr), hipMemcpyHostToDevice);

	random_device rd;
	mt19937 generator(rd());

	int epochs = 3;
	for(int j = 0; j < epochs; ++j) {
		shuffle(batches.begin(), batches.end(), generator);
		d_arr[0].unique = batches[0];
		cout << "epoch: " << j << '\n';
		for (int i = 0; i < batches.size(); ++i) {
			sample_layer<<<1,1>>>(d_graph, &d_arr[i+1], d_arr[i].unique);
			// sample_layer(&h_graph, &arr[i+1], d_arr[i].unique);
			remove_duplicates(d_arr[i+1].unique);
		}
	}
	hipFree(edges_d);
	hipFree(d_arr);
	hipFree(nodes_d);
	hipFree(sample_d);	
	hipFree(d_graph);
}