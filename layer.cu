#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>
#include <iostream>
#include <vector>
#include <fstream>
// #include <map>
// #include <algorithm>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
// #include <thrust/host_vector.h>
// #include <thrust/device_vector.h>
// #include <thrust/generate.h>
// #include <thrust/sort.h>
// #include <thrust/copy.h>
// #include <thrust/random.h>
// #include <hipcub/hipcub.hpp>

using namespace std;

typedef struct block {
	vector<int> offset;
	vector<int> values;
	vector<int> unique;
} block;

typedef struct graphStruct {
	vector<int> indptr;
	vector<int> indices;
} graphStruct;

//loop on offset as threshold
void to_graph(int offset[], int values[], int len) {	
	//	int edges[sizeof(values)/sizeof(*values)];
	for (int i = 0; i < len - 1; ++i) {
		for (int j = offset[i]; j < offset[i+1]; ++j) {
			printf("%d, %d\n", i, values[j]); 
		}
	}	
}

void to_csr(vector<vector<int>> graph) {
	// len(offset) == num_nodes + 1 (will hardcode this for now)
	int offset[5];
	int len = graph.size();
	int indices[len];
	int currN = -1;
	int ind = -1;
	// loop through to len
	for (int i = 0; i < len; ++i) {
		while (graph[i][0] != currN) {
			printf("graph value %d\n", graph[i][0]);
			currN = (int) graph[i][0];
			cout << "index: " << ind << "\n";
			offset[++ind] = i;
		}
		indices[i] = graph[i][1];
	}
	offset[++ind] = len;
	for (int i = 0; i < 5; ++i) {
		printf("offset %d: %d\n", i, offset[i]);
	}
	for (int i = 0; i < len; ++i) {
		printf("indices %d: %d\n", i, indices[i]);
	}
}
//almost done
void sample_layer(struct graphStruct* graph, struct block* t_block, vector<int> target) {
	int offset = 0;
	for (int x: target) {
		t_block->offset.push_back(offset);
		for (int i = graph->indptr[x]; i < graph->indptr[x+1]; ++i, ++offset) {
			t_block->values.push_back(graph->indices[i]);
		}
	} t_block->offset.push_back(offset);

	thrust::host_vector<int> h_vec(t_block->values);
	//transfer data to the device
	thrust::device_vector<int> d_vec = h_vec;
	//sort data on the device
	thrust::sort(d_vec.begin(), d_vec.end());
	// unique only
	thrust::device_vector<int>::iterator newLast = thrust::unique(d_vec.begin(), d_vec.end());
	// transfer data back to host
	thrust::copy(d_vec.begin(), newLast, h_vec.begin());

	for (thrust::host_vector<int>::iterator hit = h_vec.begin(); dit != h_vec.end(); ++hit) {
			// cout << *it << " ";
			t_block->unique.push_back(*hit);
	}
	for (int i = 0; i < t_block->unique.size(); ++i) {
    std::cout << t_block->unique[i] << '\n';
  }

	// t_block->unique = t_block->values;
	// sort(t_block->unique.begin(), t_block->unique.end());
	cout << "number of unique in block: " << t_block->unique.size() << '\n';
	// vector<int>::iterator ip = unique(t_block->unique.begin(), t_block->unique.begin() + t_block->unique.size());

	// t_block->unique.resize(distance(t_block->unique.begin(), ip));

	// for (ip = t_block->unique.begin(); ip != t_block->unique.end(); ++ip) {
  //   cout << *ip << '\n';
  // }
}

int main() {

	fstream f("../data/graph", ios::in);
	int num_ptrs;
	int num_edges;
	int num_sample;
	f >> num_ptrs;
	f >> num_edges;
	f >> num_sample;

	// confirmed that these are read in as ints

	fstream nodesf("../data/indptr", ios::in | ios::binary );
	if(!nodesf) {
		cout << "cannot open file!\n";
		return 0;
	}
	long *nodes_b = (long *)malloc (num_ptrs * sizeof(long));
	nodesf.read((char *)nodes_b, (num_ptrs * sizeof(long)));

	fstream edgesf("../data/indices", ios::in | ios::binary );
	if(!edgesf) {
		cout << "cannot open file!\n";
		return 0;
	}
	long *edges_b = (long *)malloc (num_edges * sizeof(long));
	edgesf.read((char *)edges_b, (num_edges * sizeof(long)));

	fstream samplef("../data/train", ios::in | ios::binary );
	if(!samplef) {
		cout << "cannot open file!\n";
		return 0;
	}
	long *sample_b = (long *)malloc (num_sample * sizeof(long));
	samplef.read((char *)sample_b, (num_sample * sizeof(long)));

	// finally done with loading the files
	// for (int i = 0; i < 10; i++) {
	// 	cout << sample_b[i] << '\n';
	// }

	// try without thrust first
	vector<vector<int>> batches;

	for (int i = 0; i < num_sample - 1024; i += 1024) {
		vector<int> batch;
		for (int j = i; j < i + 1024; ++j) {
			batch.push_back(sample_b[j]);
		}
		batches.push_back(batch);
	}
	//checking correctness
	// for (int i = 0; i < batches.size(); ++i) {
	// 	for (int j = 0; j < batches[i].size(); ++j) {
	// 		cout << batches[i][j] << '\n';
	// 		cout << sample_b[i*1024 + j] << '\n';
	// 	}
	// }

	// // b = (long *)malloc (l)
	// printf("\nsampling has started :) \n");
	vector<int> a(nodes_b, nodes_b + num_ptrs);
	vector<int> b(edges_b, edges_b + num_edges);
	graphStruct sample_graph = {a, b};

	block arr[batches.size()];
	arr[0].unique = batches[0];
	// the first unique is not unique (not sure why sample_b has duplicates)
	// for (int i = 0; i < arr[0].unique.size(); ++i) {
	// 	cout << arr[0].unique[i] << '\n';
	// }
	for (int i = 0; i < 3; ++i) {
		sample_layer(&sample_graph, &arr[i+1], arr[i].unique);
	}
	// vector<int> targetNodes();
	// arr[0].unique = targetNodes;
	// // 1024 - 4096 batch size
	// // to_csr(g);
	// // todo: fill graphStruct while reading binary
	// // what to do with graph characteristic file (for output?)
	// for (int i = 0; i < 3; ++i) {
	// 	sample_layer(&sample_graph, &arr[i+1], arr[i].unique);
	// // cout << arr[1].unique; # why cant i print this out?
	// }
}