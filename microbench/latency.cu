#include<iostream>
#include <hip/hip_runtime.h>
#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char* const func, const char* const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        // We don't exit when we encounter CUDA errors in this example.
        // std::exit(EXIT_FAILURE);
    }
}

__global__ void add(float *a, int size){
        for(int i=0;i<size;i++){
		int j = (i )%size;
		a[j] = 1;
	}

}

__global__ void add_not_zero(float *a, int size){
	for(int i=0;i<size;i++){
		int j = (i )%size;
		a[j] = 10;
	}
}

__global__ void add_trad(float *a, int size){
        for(int i=0;i<size;i++){
                int j = (i )%size;
                a[j] = 10;
        }
}


int main(){
         float *x, * y, *z;
	 int GB = 1024 * 1024 * 1024/4;
	 //Note malloc managed results in a single memcpy
         hipHostAlloc(&x, sizeof(float) * GB, 0);
	 hipMallocManaged(&y, sizeof(float) * GB);
         CHECK_CUDA_ERROR(hipMalloc(&z, sizeof(float) * GB));
	 hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	 hipEventRecord(start);
	 for(int i=0;i<10;i++){
	 		add<<<1,1>>>(x,GB);
		//	add_not_zero<<<1,1>>>(y, GB);
	 	//	add_trad<<<1,1>>>(z, GB);
	 }
	 CHECK_CUDA_ERROR(hipEventRecord(stop));
         CHECK_CUDA_ERROR(hipEventSynchronize(stop));
	 float milliseconds = 0;
	 hipEventElapsedTime(&milliseconds, start, stop);
	 std::cout << GB << " ";
	 std::cout << "time :" << milliseconds <<"\n";
         std::cout << "Hello world\n";
}

