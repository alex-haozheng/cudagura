
#include <hip/hip_runtime.h>
#include<iostream>



__global__ void add(float *a){
	
	a[0] = 998.8;
	
}

int main(){
	 float *x;
 	 hipMallocManaged(&x, sizeof(float));	
	 add<<<1,1>>>(x);
	 hipDeviceSynchronize();
	 std::cout << x[0] <<"\n";
	 std::cout << "Hello world\n";
}
