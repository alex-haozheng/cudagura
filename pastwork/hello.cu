
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main() {
    hipSetDevice(0);
    cuda_hello<<<1,1>>>(); 
    hipDeviceSynchronize();
    return 0;
}

/*


*/