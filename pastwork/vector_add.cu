#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#define N 100000


using namespace std;

__global__ void vector_add(float *out, float *a, float *b, int n) {
    for (int i = 0; i < n; i++) {
        out[i] = a[i] + b[i];
    }
}

int main() {
    float *a, *b, *out;
    float *d_a, *d_b, *d_out;
    hipEvent_t start, stop;
    float elapsed_time;
    // allocate host memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    hipEventCreate(&start);
    hipEventCreate(&stop);
    // initialize array on host
    for(int i = 0; i < N; i++) {
        a[i] = i; b[i] = i;
    }
    // allocate device memory
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    hipEventRecord(start, 0);
    // Main function
    vector_add<<<1,1>>>(d_out, d_a, d_b, N);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);

    cout << elapsed_time << '\n';
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
    
    // for (int i = 0; i < N; ++i) {
   	//     printf("a: %f, b: %f, o: %f\n", a[i], b[i], out[i]);
    // }
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
}