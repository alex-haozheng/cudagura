
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#define N 100000

__global__ void vector_add(float *out, float *a, float *b, int n) {
    for (int i = 0; i < n; i++) {
        out[i] = a[i] + b[i];
    }
}

int main() {
    float *a, *b, *out;
    float *d_a, *d_b, *d_out;

    // allocate host memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // allocate device memory
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // initialize array on device
    for(int i = 0; i < N; i++) {
        a[i] = 1.0f; b[i] = 2.0f;
    } // redundancy?

    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
    // Main function
    vector_add<<<1,1>>>(d_out, d_a, d_b, N);

    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
       
    for (int i = 0; i < 10; ++i) {
   	    printf("a: %f, b: %f, o: %f\n", a[i], b[i], out[i]);
    }
}

