#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdbool.h>
#include <stdlib.h>
#include <iostream>
#include <vector>
#include <fstream>
#include <random>
#include <chrono>
#include <algorithm>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "nvtx3/nvToolsExt.h"
#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

using namespace std;

typedef struct block {
	// thrust::device_vector
	thrust::device_vector<long> offset;
	thrust::device_vector<long> indices;
	thrust::device_vector<long> unique;
	// thrust::device_vector<long> in_degrees;

	void clear(){
		offset.clear();
		indices.clear();
	}
} block;

// typedef struct graphStruct {
// 	thrust::device_vector<long> indptr;
// 	thrust::device_vector<long> indices;
// } graphStruct;


const int THREAD_SIZE = 256;
const int MAX_BLOCKS = 1024;
int BLOCK_SIZE(size_t t){
	if(t > MAX_BLOCKS) return MAX_BLOCKS;
	return (t-1)/THREAD_SIZE + 1;
}

__global__ void init_random_states(hiprandState *states, size_t num,
                                   unsigned long seed) {
  size_t threadId = threadIdx.x + blockIdx.x * blockDim.x;
  assert(num == blockDim.x * gridDim.x);
  if (threadId < num) {
    // Copied from GNNLAB
    /** Using different seed & constant sequence 0 can reduce memory
      * consumption by 800M
      * https://docs.nvidia.com/cuda/hiprand/device-api-overview.html#performance-notes
      */
    hiprand_init(seed+threadId, 0, 0, &states[threadId]);
  }
}

void remove_duplicates(thrust::device_vector<long>& nodes){
  if(nodes.size() == 0)return;
  if(nodes.size() > 1){
    nvtxRangePush("remove duplicates"); // built in timing?
    thrust::sort(nodes.begin(), nodes.end());
    auto it = thrust::unique(nodes.begin(), nodes.end());
    nodes.erase(it, nodes.end());
    nvtxRangePop();
  }
}

__global__
void printing(long* indptr_g) {
	for (int i = 0; i < 10; ++i){
		printf("index: %d value: %ld \n", i, indptr_g[i]);
	}
}

__global__
void sample_offsets(long *in, size_t in_size, long *offsets_s, 
	long *indptr_g, long num_nodes, int fanout){
      int id = blockIdx.x * blockDim.x + threadIdx.x;
  while(id < in_size){
      int nd = in[id];
			// get # of neighbors
      int nbs_size = indptr_g[nd+1] - indptr_g[nd];
			
			// stabilize nbs_size
			if(fanout < nbs_size){
					nbs_size = fanout;
			}
      
      offsets_s[id+1] = nbs_size;
      id = id + (blockDim.x * gridDim.x);
  }
}

__global__
void neigh_sample(long * in, long size, long * offsets, long * indices,\
      long * graph_indptr, long * graph_indices, long num_nodes, int fanout, hiprandState *random_states) {
      int threadId =  blockIdx.x * blockDim.x + threadIdx.x;
      int id = threadId;
      while(id < size){
          int nd = in[id];
          int nbs_size = graph_indptr[nd+1] - graph_indptr[nd];
					// sets the starting position to start reading in indices (graph standpoint)
          long *read = &graph_indices[graph_indptr[nd]];
					// where to start writing (from indices standpoint)
          long *write = &indices[offsets[id]];
          if(nbs_size > fanout){
						for(int j = 0; j < fanout; j++){
							int sid = (int) (hiprand_uniform(&random_states[threadId]) * nbs_size);
            	write[j] = read[sid];
						}
          }else{
						for(int j = 0; j < nbs_size; j++){
							write[j] = read[j];
						}
          }
          id = id + (blockDim.x * gridDim.x);
      }
  }

// todo: update the variables witin the functions 
void sample_layer(long *g_indptr, long *g_indices, thrust::device_vector<long> &target, 
	thrust::device_vector<long> &offsets, thrust::device_vector<long> &indices, long fanout, long num_nodes, hiprandState *random_states) {
		offsets.clear();
		indices.clear();
		offsets.resize(target.size() + 1);
		// appending for inclusive scan
		// cout << "before offset sampling\n";
		offsets[0] = 0;
		sample_offsets<<<BLOCK_SIZE(target.size()), THREAD_SIZE>>>
			(thrust::raw_pointer_cast(target.data()), target.size(), \
    		thrust::raw_pointer_cast(offsets.data()), \
      		g_indptr, num_nodes, fanout);
		// cout << "after sample offset\n";
		thrust::inclusive_scan(thrust::device, offsets.begin(), 
			offsets.end(), offsets.begin()); 
		indices.resize(offsets[offsets.size() - 1]);
		// cout << "segfault in sample layer\n";
		neigh_sample<<<BLOCK_SIZE(target.size()), THREAD_SIZE>>>
			(thrust::raw_pointer_cast(target.data()), target.size(),
            thrust::raw_pointer_cast(offsets.data()),
             thrust::raw_pointer_cast(indices.data()),
              g_indptr, g_indices, num_nodes, fanout, random_states);
	}

int main() {
	// reading the file shouldn't be just move the data read into device
	fstream f("../data/graph", ios::in);
	long num_ptrs;
	long num_edges;
	long num_sample;
	f >> num_ptrs;
	f >> num_edges;
	f >> num_sample;

	// try these diff types
	// malloc
	// mallocmanaged
	// mallocHostAlloc
	fstream nodesf("../data/indptr", ios::in | ios::binary );
	if(!nodesf) {
		cout << "cannot open file!\n";
		return 0;
	}
	long *nodes_h = (long *)malloc (num_ptrs * sizeof(long));
	nodesf.read((char *)nodes_h, (num_ptrs * sizeof(long)));

	// for (int i = 0; i < 10; ++i){
	// 	printf("index: %d cpu value: %ld \n", i, nodes_h[i]);
	// }

	fstream edgesf("../data/indices", ios::in | ios::binary );
	if(!edgesf) {
		cout << "cannot open file!\n";
		return 0;
	}

	long *edges_h = (long *)malloc (num_edges * sizeof(long));
	edgesf.read((char *)edges_h, (num_edges * sizeof(long)));

	fstream samplef("../data/train", ios::in | ios::binary );
	if(!samplef) {
		cout << "cannot open file!\n";
		return 0;
	}
	long *sample_h = (long *)malloc (num_sample * sizeof(long));
	samplef.read((char *)sample_h, (num_sample * sizeof(long)));

	vector<vector<long>> batches;

	for (int i = 0; i < num_sample - 1024; i += 1024) {
		vector<long> batch;
		for (int j = i; j < i + 1024; ++j) {
			batch.push_back(sample_h[j]);
		}
		batches.push_back(batch);
	}
	// end
	vector<long> a(nodes_h, nodes_h + num_ptrs);
	vector<long> b(edges_h, edges_h + num_edges);	
	long *nodes_g;
	long *edges_g;
	// malloc
	// mallocmanaged
	// mallocHostAlloc
	// gpuErrchk(hipMalloc((void**)&nodes_g, (num_ptrs - 1) * sizeof(long)));
	// gpuErrchk(hipMemcpy(nodes_g, nodes_h, (num_ptrs - 1) * sizeof(long), hipMemcpyHostToDevice));
	// gpuErrchk(hipMalloc((void**)&edges_g, (num_edges - 1) * sizeof(long)));
	// gpuErrchk(hipMemcpy(edges_g, edges_h, (num_edges - 1) * sizeof(long), hipMemcpyHostToDevice));
	hipMallocManaged(&nodes_g, (num_ptrs - 1) * sizeof(long));
	hipMallocManaged(&edges_g, (num_edges - 1) * sizeof(long));
	// hipHostMalloc((void**)&nodes_g, (num_ptrs - 1) * sizeof(long));
	// hipHostMalloc((void**)&edges_g, (num_edges - 1) * sizeof(long));

	
	const int TOTAL_RAND_STATES = 1024 * 256;
	hiprandState* dev_curand_states;
	unsigned long seed = std::chrono::system_clock::now().time_since_epoch().count();
	gpuErrchk(hipMalloc(&dev_curand_states, TOTAL_RAND_STATES * sizeof(hiprandState)));
	init_random_states<<<MAX_BLOCKS, THREAD_SIZE>>>(dev_curand_states, TOTAL_RAND_STATES, seed);
	
	
	block **d_arr = (block **) malloc(sizeof(block) * 3);
	for (int i=0;i<3;++i) {
		d_arr[i] = new block();
	}

	// gpuErrchk(hipMalloc((void **) &d_arr, sizeof(arr)));
	// gpuErrchk(hipMemcpy(d_arr, &arr, sizeof(arr), hipMemcpyHostToDevice));

	random_device rd;
	mt19937 generator(rd());
	float elapsed_time;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int epochs = 3;
	hipEventRecord(start, 0);
	for(int j = 0; j < epochs; ++j) {
		shuffle(batches.begin(), batches.end(), generator);
		for (int k = 0; k < batches.size(); ++k) {
			thrust::device_vector<long> b(batches[k].begin(), batches[k].end());
			d_arr[0]->unique = b;
			for (int i = 1; i <= 3; ++i) {
				// still passing in wrong pointer
				sample_layer(nodes_g, edges_g, d_arr[i-1]->unique, d_arr[i]->offset, d_arr[i]->indices, 20, num_ptrs, dev_curand_states);
				// for (int c = 0; c < d_arr[i]->indices.size(); ++c) {
				// 	cout << d_arr[i]->indices[c] << '\n';
				// }
				d_arr[i]->unique.resize(d_arr[i]->indices.size());
				thrust::copy(thrust::device, d_arr[i]->indices.begin(), d_arr[i]->indices.end(), d_arr[i]->unique.begin());
				remove_duplicates(d_arr[i]->unique);
				for (int i=0;i<=3;++i) {
					d_arr[i] = new block();
				}
			}
		}
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time, start, stop);
	cout << elapsed_time << '\n';	
	hipEventDestroy(start);
  hipEventDestroy(stop);
	free(d_arr);
}