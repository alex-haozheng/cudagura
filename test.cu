#include <hip/hip_runtime.h>
#include <iostream>

struct my_struct {
  int x;
  int y;
};

__global__ void process_struct(struct my_struct *d_struct) {
  // Access data in the struct on the GPU
  int x = d_struct->x;
  int y = d_struct->y;

  // Do something with the data...

  // Update the struct data on the GPU
  d_struct->x = x + 1;
  d_struct->y = y + 2;
}

int main() {
  struct my_struct h_struct;
  struct my_struct *d_struct;

  // Initialize data on the CPU
  h_struct.x = 1;
  h_struct.y = 2;

  // Allocate memory on the device
  hipMalloc((void**)&d_struct, sizeof(struct my_struct));

  // Transfer data from host to device
  hipMemcpy(d_struct, &h_struct, sizeof(struct my_struct), hipMemcpyHostToDevice);

  // Launch a kernel to process the struct on the GPU
  process_struct<<<1, 1>>>(d_struct);

  // Transfer data from device to host
  hipMemcpy(&h_struct, d_struct, sizeof(struct my_struct), hipMemcpyDeviceToHost);

  // Verify the data on the host
  std::cout << h_struct.x << " " << h_struct.y << std::endl;

  // Free memory on the device
  hipFree(d_struct);

  return 0;
}
