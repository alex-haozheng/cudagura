
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#define N 10000000

__global__ void vector_add(float *out, float *a, float *b, int n) {
    for (int i = 0; i < n; i++) {
        out[i] = a[i] + b[i];
    }
}

int main() {
    float *a, *b, *out;
    float *d_a;

    // allocate device memory for a
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++) {
        a[i] = 1.0f; b[i] = 2.0f;
    }

    // Main function
    vector_add<<<1,1>>>(out, a, b, N);

    for (int i = 0; i < 10; ++i) {
   	    printf("a: %f, b: %f, o: %f\n", a[i], b[i], out[i]);
    }
}

